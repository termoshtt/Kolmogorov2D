
#include "Converter.hpp"

namespace Kolmogorov2D {

template <> ConverterC2R<float>::ConverterC2R(int Nx, int Ny) {
  cujak::exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_C2R));
}
template <> ConverterC2R<double>::ConverterC2R(int Nx, int Ny) {
  cujak::exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_Z2D));
}

template <>
void ConverterC2R<float>::operator()(const Complex *uf, Real *u) const {
  cujak::exec(hipfftExecC2R(plan, const_cast<Complex *>(uf), u));
}

template <>
void ConverterC2R<double>::operator()(const Complex *uf, Real *u) const {
  cujak::exec(hipfftExecZ2D(plan, const_cast<Complex *>(uf), u));
}

template <> ConverterR2C<float>::ConverterR2C(int Nx, int Ny) {
  cujak::exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_R2C));
}
template <> ConverterR2C<double>::ConverterR2C(int Nx, int Ny) {
  cujak::exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_D2Z));
}

template <>
void ConverterR2C<float>::operator()(const Real*u, Complex *uf) const {
  cujak::exec(hipfftExecR2C(plan, const_cast<Real *>(u), uf));
}

template <>
void ConverterR2C<double>::operator()(const Real *u, Complex *uf) const {
  cujak::exec(hipfftExecD2Z(plan, const_cast<Real *>(u), uf));
}

} // namespace Kolmogorov2D
