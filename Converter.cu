
#include "Converter.hpp"

namespace Kolmogorov2D {

template <> ConverterC2R<float>::ConverterC2R(int Nx, int Ny) {
  cujak::exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_C2R));
}
template <> ConverterC2R<double>::ConverterC2R(int Nx, int Ny) {
  cujak::exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_Z2D));
}

template <>
void ConverterC2R<float>::operator()(const Complex *uf, Real *u) const {
  cujak::exec(hipfftExecC2R(plan, const_cast<Complex *>(uf), u));
}

template <>
void ConverterC2R<double>::operator()(const Complex *uf, Real *u) const {
  cujak::exec(hipfftExecZ2D(plan, const_cast<Complex *>(uf), u));
}

} // namespace Kolmogorov2D
